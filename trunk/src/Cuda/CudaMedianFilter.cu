#include "hip/hip_runtime.h"
/*
* Cuda Library.
*
* Copyright (c) 2011-2014 Yermalayeu Ihar.
*
* Permission is hereby granted, free of charge, to any person obtaining a copy 
* of this software and associated documentation files (the "Software"), to deal
* in the Software without restriction, including without limitation the rights
* to use, copy, modify, merge, publish, distribute, sublicense, and/or sell 
* copies of the Software, and to permit persons to whom the Software is 
* furnished to do so, subject to the following conditions:
*
* The above copyright notice and this permission notice shall be included in 
* all copies or substantial portions of the Software.
*
* THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR 
* IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY, 
* FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE 
* AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER 
* LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
* OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
* SOFTWARE.
*/
//#include "Simd/SimdMath.h"

#include "Cuda/CudaMath.h"
#include "Cuda/CudaAlg.h"
#include <hip/device_functions.h>

namespace Cuda
{
    template <class T> CUDA_INLINE __device__ T Max(T a, T b)
    {
        return a > b ? a : b;
    }

    template <class T> CUDA_INLINE __device__ T Min(T a, T b)
    {
        return a < b ? a : b;
    }

    template <class T> CUDA_INLINE __device__ void Sort(T & a, T & b)
    {
        if(a > b)
        {
            T t = a;
            a = b;
            b = t;
        }
    }

    template <class T> CUDA_INLINE __device__ void LoadRhomb3x3(const uint8_t * y[3], size_t x[3], size_t c, T a[5])
    {
        a[0] = y[0][x[1] + c];
        a[1] = y[1][x[0] + c]; a[2] = y[1][x[1] + c]; a[3] = y[1][x[2] + c];
        a[4] = y[2][x[1] + c]; 
    }

    template <class T> CUDA_INLINE __device__ void PartialSort5(T a[5])
    {
        Sort(a[2], a[3]); 
        Sort(a[1], a[2]);
        Sort(a[2], a[3]); 
        a[4] = Max(a[1], a[4]); 
        a[0] = Min(a[0], a[3]); 
        Sort(a[2], a[0]); 
        a[2] = Max(a[4], a[2]); 
        a[2] = Min(a[2], a[0]);
    }

    __global__ void MedianFilterRhomb3x3Kernel(const uint8_t * src, size_t srcStride, size_t width, size_t height, 
        size_t channelCount, uint8_t * dst, size_t dstStride)
    {
        uint8_t a[5];
        const uint8_t * y[3];
        size_t x[3];

        const int row = blockDim.x * blockIdx.x + threadIdx.x;
        const int col = blockDim.y * blockIdx.y + threadIdx.y;

        if(row >= height || col >= width)
            return;    

        y[0] = src + srcStride*Max(row - 1, 0);
        y[1] = src + srcStride*row;
        y[2] = src + srcStride*Min<int>(row + 1, height - 1);

        x[0] = channelCount*Max(col - 1, 0);
        x[1] = channelCount*col;
        x[2] = channelCount*Min<int>(col + 1, width - 1);

        #pragma unroll
        for(size_t channel = 0; channel < channelCount; channel++)
        {
            LoadRhomb3x3(y, x, channel, a);
            PartialSort5(a);
            dst[row*dstStride + col*channelCount + channel] = (uint8_t)a[2];
        }
    }

    void MedianFilterRhomb3x3(const uint8_t * src, size_t srcStride, size_t width, size_t height, 
        size_t channelCount, uint8_t * dst, size_t dstStride)
    {
        const int n = 1;
        const int m = 128;
        const ::dim3 grid((height + n - 1)/n, (width + m - 1)/m, 1);      
        const ::dim3 block(n, m, 1); 
       
        MedianFilterRhomb3x3Kernel<<<grid, block>>>(src, srcStride, width, height, channelCount, dst, dstStride);
        ::hipDeviceSynchronize();
    }
}